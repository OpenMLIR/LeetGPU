#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // Column index
    int y = blockIdx.y * blockDim.y + threadIdx.y;  // Row index

    if (x < cols && y < rows) {
        output[x * rows + y] = input[y * cols + x]; // Transpose operation
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int rows, int cols) {
    const int BLOCK_SIZE = 16;
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE,
                       (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}
