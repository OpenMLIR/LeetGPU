#include <hip/hip_runtime.h>

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // Column index
    int y = blockIdx.y * blockDim.y + threadIdx.y;  // Row index

    if (x < cols && y < rows) {
        output[x * rows + y] = input[y * cols + x]; // Transpose operation
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}
